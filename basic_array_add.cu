#include <hip/hip_runtime.h>
#include <iostream>

// Kernel: out[x] = a[x] + b[x]
__global__ void add(int n, const int* a, const int* b, int* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int N = 1024;
    size_t size = N * sizeof(int);

    int h_a[N], h_b[N], h_c[N];
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int  bsize = 256;
    dim3 blockSize(bsize);
    dim3 gridSize((N + bsize - 1) / bsize);

    add<<<blockSize, gridSize>>>(N, d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << "\n";
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
